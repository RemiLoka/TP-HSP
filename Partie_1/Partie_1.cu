#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

void MatrixInit(float *M, int n, int p);
void MatrixPrint(float *M, int n, int p);
void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p);
void MatrixMult(float *M1, float *M2, float *Mout, int n);

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p);
__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n);



void MatrixInit(float *M, int n, int p) {
    for (int i = 0; i < n * p; i++) {
        M[i] = ((float)rand() / (float)(RAND_MAX)) * 2.0 - 1.0;
    }
}

void MatrixPrint(float *M, int n, int p) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            printf("%f ", M[i * p + j]);
        }
        printf("\n");
    }
}


void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    for (int i = 0; i < n * p; i++) {
        Mout[i] = M1[i] + M2[i];
    }
}

void MatrixMult(float *M1, float *M2, float *Mout, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++){
            float sum = 0;
            for (int k = 0; k < n; k++) {
                sum += M1[i * n + k] * M2[k * n + j];
            }
            Mout[i * n + j] = sum;
        }
    }
}


__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < p) {
        Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
    }
}


__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        float sum = 0;
        for (int k = 0; k < n; k++) {
            sum += M1[i * n + k] * M2[k * n + j];
        }
        Mout[i * n + j] = sum;
    }
}


int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: %s n p\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int p = atoi(argv[2]);
    size_t size = n * p * sizeof(float);


    float *h_M1 = (float *)malloc(size);
    float *h_M2 = (float *)malloc(size);
    float *h_Mout = (float *)malloc(size);
    MatrixInit(h_M1, n, p);
    MatrixInit(h_M2, n, p);

    
    clock_t start_cpu, end_cpu;
    start_cpu = clock();
    MatrixAdd(h_M1, h_M2, h_Mout, n, p);
    end_cpu = clock();
    double cpu_time_used = ((double) (end_cpu - start_cpu)) / CLOCKS_PER_SEC;
    printf("Temps CPU ADD : %f secondes\n", cpu_time_used);

    clock_t start_cpu2, end_cpu2;
    start_cpu2 = clock();
    MatrixMult(h_M1, h_M2, h_Mout, n);
    end_cpu2 = clock();
    double cpu_time_used2 = ((double) (end_cpu2 - start_cpu2)) / CLOCKS_PER_SEC;
    printf("Temps CPU MULT : %f secondes\n", cpu_time_used2);

    
    float *d_M1, *d_M2, *d_Mout;
    hipMalloc((void **)&d_M1, size);
    hipMalloc((void **)&d_M2, size);
    hipMalloc((void **)&d_Mout, size);

    
    hipMemcpy(d_M1, h_M1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_M2, h_M2, size, hipMemcpyHostToDevice);

    
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (p + threadsPerBlock.y - 1) / threadsPerBlock.y);

    
    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);
    hipEventRecord(start_gpu);
    cudaMatrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_M1, d_M2, d_Mout, n, p);
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_gpu, stop_gpu);
    printf("Temps GPU ADD: %f millisecondes\n", milliseconds);

    hipEvent_t start_gpu2, stop_gpu2;
    hipEventCreate(&start_gpu2);
    hipEventCreate(&stop_gpu2);
    hipEventRecord(start_gpu2);
    cudaMatrixMult<<<blocksPerGrid, threadsPerBlock>>>(d_M1, d_M2, d_Mout, n);
    hipEventRecord(stop_gpu2);
    hipEventSynchronize(stop_gpu2);
    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start_gpu2, stop_gpu2);
    printf("Temps GPU MULT: %f millisecondes\n", milliseconds2);


    hipMemcpy(h_Mout, d_Mout, size, hipMemcpyDeviceToHost);
    //MatrixPrint(h_Mout, n, p);

    
    free(h_M1);
    free(h_M2);
    free(h_Mout);
    hipFree(d_M1);
    hipFree(d_M2);
    hipFree(d_Mout);

    return 0;
}
